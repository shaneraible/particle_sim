#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define MAX_BIN_SIZE 15
extern double size;

typedef struct {
    particle_t particles[MAX_BIN_SIZE];
    int size;
} bin_t;

//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__device__ void rebin_gpu(particle_t * particles, bin_t * bins, int n, int bins_per_dim){

}

__global__ void compute_forces_gpu(particle_t * particles, bin_t * bins, int n, int bins_per_dim, int size)
{
    // Get thread (particle) ID
    int thread = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread >= n) return;

    int i = thread / bins_per_dim;
    int j = thread % bins_per_dim;

    int block_num = i*bins_per_dim+j;
    bin_t &current_bin = bins[block_num];

    for (int k = 0; k < current_bin.size; k++){
        current_bin.particles[k].ax = current_bin.particles[k].ay = 0;
    }

    
    // zero out acceleration
    // current_bin.particles[particle_idx].ax = current_bin.particles[particle_idx].ay = 0;
    // for (int i = 0; i < current_bin.size; i++){
    //     int bin_r = round(double(particles[i].y)/size*(bins_per_dim-1));
    //     int bin_c = round(double(particles[i].x)/size*(bins_per_dim-1));

    //     for(int r = max(bin_r - 1, 0); r <= min(bin_r+1, bins_per_dim - 1); r ++)
    //     {
    //         for(int c = max(bin_c - 1, 0); c <= min(bin_c+1, bins_per_dim - 1); c++)
    //         {
    //             bin_t &neighbor = bins[r*bins_per_dim + c];
    //             //forces within this bin
    //             for (int j = 0; j < neighbor.size; j++){
    //                 if(threadIdx.x!=j) apply_force_gpu(current_bin.particles[i], neighbor.particles[j]);
    //             }
    //         }
    //     }
    // } 

    // //forces within this bin
    for (int i = 0; i < current_bin.size; i++){
      for (int j = 0; j < current_bin.size; j++){
        if(threadIdx.x!=j) apply_force_gpu(current_bin.particles[i], current_bin.particles[j]);
      }
    }   

    // for(int j = 0 ; j < n ; j++)
    //     apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

void init_bins(bin_t *bins, particle_t* particles, int n, int bins_per_dim, double grid_size){
    for(int i=0; i<bins_per_dim*bins_per_dim; i++){

        bins[i].size = 0;
    }

        //store the point into the grid
    for (int i = 0; i < n; i++){
        int x = min((int)(particles[i].x / (grid_size/bins_per_dim)), bins_per_dim - 1);
        int y = min((int)(particles[i].y / (grid_size/bins_per_dim)), bins_per_dim - 1);
        int index = y*bins_per_dim + x;
    
        if(bins[index].size >= MAX_BIN_SIZE){
            printf("BIN SIZE TOO SMALL\n");
            return;
        }
        bins[index].particles[bins[index].size++] = particles[i];

    }
}

int main( int argc, char **argv )
{    

    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    set_size( n );
    init_particles( n, particles );

    double size = get_size();
    int bins_per_dim = int(sqrt(n/4));

    bin_t *bins = (bin_t*) malloc(bins_per_dim*bins_per_dim*sizeof(bin_t));
    init_bins(bins, particles, n, bins_per_dim, size);
    
    bin_t *bins_gpu; 
    hipMalloc((void **) &bins_gpu, bins_per_dim * bins_per_dim * sizeof(bin_t));

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));


    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
    hipMemcpy(bins_gpu, bins, bins_per_dim * bins_per_dim * sizeof(bin_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

        int blks = (bins_per_dim*bins_per_dim + NUM_THREADS - 1)/NUM_THREADS;
        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, bins_gpu, n, bins_per_dim, size);
        
        //
        //  move particles
        //
	    move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	        // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(bins, bins_gpu, bins_per_dim*bins_per_dim*sizeof(bin_t), hipMemcpyDeviceToHost);
            int count = 0;
            for (int p = 0; p < bins_per_dim; p++){
                for (int q = 0; q < bins_per_dim; q++){
                    bin_t& curr = bins[p*bins_per_dim + q];
                    for (int k = 0; k < curr.size; k++){
                        particles[count++] = curr.particles[k];
                    }
                }
            }

            save( fsave, n, particles);
            hipMemset(bins_gpu, 0, bins_per_dim * bins_per_dim * sizeof(bin_t));
            init_bins(bins, particles, n, bins_per_dim, size);
            hipMemcpy(bins_gpu, bins, bins_per_dim * bins_per_dim * sizeof(bin_t), hipMemcpyHostToDevice);

	    }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
